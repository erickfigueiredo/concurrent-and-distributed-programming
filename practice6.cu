#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <chrono>
#include <iostream>

#define BLOCK_SIZE 32

using namespace std;

/* A função CHECK a seguir pode ser utilizada caso seja necessário identificar
erros em chamadas da biblioteca CUDA. Exemplo: 
CHECK(cudaMalloc((void**)&x,size));
*/
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

__global__ void calcVizinhos(int8_t *m, int linhas, int colunas){
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    int j = blockIdx.x*blockDim.x + threadIdx.x;

    if(i >= linhas || j >= colunas || m[i*colunas + j] == 9) return;

    int vizinhos[][2] = {{0,1},{1,1}, {-1,1}, {-1,0}, {1,0}, {0,-1},{1,-1}, {-1,-1}};
    int auxX = 0;
    int auxY = 0;

    for(int k = 0; k < 8; k++) {
      auxX = i+vizinhos[k][0];
      auxY = j+vizinhos[k][1];

      if(auxX >=0 && auxX < linhas && auxY >=0 && auxY < colunas && m[auxX*colunas + auxY] == 9) {
          m[i*colunas + j] ++;
      }
    }
}

void calcVizinhosSerial(int8_t *m, int linhas, int colunas){
  int vizinhos[][2] = {{0,1},{1,1}, {-1,1}, {-1,0}, {1,0}, {0,-1},{1,-1}, {-1,-1}};
  int auxX = 0, auxY = 0;

  for(int i = 0; i < linhas; i++){   
    for(int j = 0; j < colunas; j++){
        if(m[i*colunas+j] == 9) continue;

        for(int k = 0; k < 8; k++) {
          auxX = i+vizinhos[k][0];
          auxY = j+vizinhos[k][1];

          if(auxX >=0 && auxX < linhas && auxY >=0 && auxY < colunas && m[auxX*colunas + auxY] == 9)
            m[i*colunas + j] ++;
        }
    }
  }
}

void distribuiBombas(int8_t *a, int linhas, int colunas, int num){
    if (num > linhas*colunas)
        throw std::invalid_argument("Número de bombas extrapolou o máximo possível.");

    for (int i=0; i<linhas; ++i)
        for (int j=0; j<colunas; ++j)
            a[i*colunas+j]=0;

    int lin, col;
    for (int i=0; i<num; ++i) {
        lin=rand()%linhas;
        col=rand()%colunas;
        while (a[lin*colunas+col] == 9) {
            lin=rand()%linhas;
            col=rand()%colunas;
        }
        a[lin*colunas+col]=9;
    }
}

void imprimeMatriz(int8_t *a, int linhas, int colunas ){
    int v;
    for (int i=0; i<linhas; ++i) {
        for (int j=0; j<colunas; ++j) {
            v=a[i*colunas+j];
            if (v==9) printf("* ");
            else printf("%d ",v);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
    int linha = 5000, col = 5000;
    bool calcOnGPU = true;

    int8_t *m = (int8_t *) malloc(linha*col*sizeof(int8_t));

    distribuiBombas(m, linha, col, 5);

    //imprimeMatriz(m, linha, col);

    if(calcOnGPU) {
      cout << "ON GPU\n";
      int8_t *mC;
      hipMalloc((void **) &mC, linha*col*sizeof(int8_t));

      hipMemcpy(mC, m,  linha*col*sizeof(int8_t), hipMemcpyHostToDevice);
      dim3 grid (ceil(col*1.0/BLOCK_SIZE),ceil(linha*1.0/BLOCK_SIZE),1);
      dim3 block (BLOCK_SIZE,BLOCK_SIZE,1);

      calcVizinhos <<<grid, block>>>(mC, linha, col);

      hipMemcpy(m, mC, linha*col*sizeof(int8_t), hipMemcpyDeviceToHost);
      
      //imprimeMatriz(m, linha, col);  
      
      hipFree(mC);
      hipDeviceReset();
    } else {
      cout << "ON CPU\n";
      auto start = chrono::system_clock::now();
      calcVizinhosSerial (m, linha, col);
      auto end = chrono::system_clock::now();
      chrono::duration<double> time = end - start;

      printf("Tempo de Execução: %.10f\n", time.count());
      //imprimeMatriz(m, linha, col);  
    }

    free(m);

    return 0;    
}
